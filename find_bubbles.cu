#include "hip/hip_runtime.h"
#define BLOCK_SIZE 8

// Convert 3D index to 1D index.
#define INDEX(k,j,i,ld) ((k)*ld*ld + (j) * ld + (i))

// Texture memory for image.
texture<float,3> img;

// Step 1. Label local minima or flatland as PLATEAU
__global__ void real_tophat_kernel(float* ionized, const int w, float R, float S0)
{
  int tx = threadIdx.x;  int ty = threadIdx.y; int tz = threadIdx.z;
  int bx = blockIdx.x;   int by = blockIdx.y; int bz = blockIdx.z;
  int bdx = blockDim.x;  int bdy = blockDim.y; int bdz = blockDim.z;
  int i = bdx * bx + tx; int j = bdy * by + ty; int k = bdz * bz + tz;
  int p = INDEX(k,j,i,w);
  if (j >= w || i >= w || k >= w || ionized[p] == 1) return;

  float rsq;
  float deltasum = 0;
  float deltac = %(DELTAC)s;
  float smin = %(SMIN)s;
  int count = 0;
  for (int kk = 0; kk < w; kk++) {
  	for (int jj = 0; jj < w; jj++) {
  		for (int ii = 0; ii < w; ii++){
  			rsq = (ii-i)*(ii-i)+(jj-j)*(jj-j)+(kk-k)*(kk-k);
  			if (rsq < R*R)
  			{
  				deltasum += tex3D(img,i,j,k);
  				count ++;
  			}
  		}
  	}
  }
  float delta0 = deltasum/count;
  float fcoll = 1 - erf((deltac - delta0)/sqrt(2*(smin - S0)));
  //ionized[p] = fcoll* %(ZETA)s;;
  if (fcoll >= 1/%(ZETA)s) ionized[p] = 1.0;
  else { ionized[p] = fcoll * %(ZETA)s; }
 }
 __global__ void k_tophat_kernel(float* ionized, const int w, float R, float S0)
{
	int tx = threadIdx.x;  int ty = threadIdx.y; int tz = threadIdx.z;
	int bx = blockIdx.x;   int by = blockIdx.y; int bz = blockIdx.z;
	int bdx = blockDim.x;  int bdy = blockDim.y; int bdz = blockDim.z;
	int i = bdx * bx + tx; int j = bdy * by + ty; int k = bdz * bz + tz;
	int p = INDEX(k,j,i,w);
	float ks = pow((9*3.14159/2),1.0/3) / R;
	if (j >= w || i >= w || k >= w || ionized[p] == 1) return;

	float rsq, r, y;
	float deltasum = 0;
	float deltac = %(DELTAC)s;
	float smin = %(SMIN)s;
	float count = 0;
	float W;
	for (int kk = 0; kk < w; kk++) {
		for (int jj = 0; jj < w; jj++) {
			for (int ii = 0; ii < w; ii++){
				rsq = (ii-i)*(ii-i)+(jj-j)*(jj-j)+(kk-k)*(kk-k);
				r = pow( rsq, float(0.5) );
				y = ks*r;
				W = (sin(y) - y*cos(y))/( 2*pow(3.14159,2)*pow(r,3) );
				deltasum += W * tex3D(img,i,j,k);
				count += W;
			}
		}
	}
	float delta0 = deltasum/count;
	float fcoll = 1 - erf((deltac - delta0)/sqrt(2*(smin - S0)));
	//ionized[p] = fcoll* %(ZETA)s;;
	if (fcoll >= 1/%(ZETA)s) ionized[p] = 1.0;
	else { ionized[p] = fcoll * %(ZETA)s; }
 }


__global__ void real_tophat(float* filter, int w, float R)
{
	int tx = threadIdx.x;  int ty = threadIdx.y; int tz = threadIdx.z;
	int bx = blockIdx.x;   int by = blockIdx.y; int bz = blockIdx.z;
	int bdx = blockDim.x;  int bdy = blockDim.y; int bdz = blockDim.z;
	int i = bdx * bx + tx; int j = bdy * by + ty; int k = bdz * bz + tz;
	int p = INDEX(k,j,i,w);
	if (j >= w || i >= w || k >= w) return;
	float rsq = (w/2-i)*(w/2-i)+(w/2-j)*(w/2-j)+(w/2-k)*(w/2-k);
	float vol = 4.0*3.1415926*R*R*R/3.0;
	if (rsq < R*R)
	{
		filter[p] = 1./vol;
	}
	else
	{
		filter[p] = 0;
	}
 }

 __global__ void k_tophat(float* filter, int w, float ks)
{
	int tx = threadIdx.x;  int ty = threadIdx.y; int tz = threadIdx.z;
	int bx = blockIdx.x;   int by = blockIdx.y; int bz = blockIdx.z;
	int bdx = blockDim.x;  int bdy = blockDim.y; int bdz = blockDim.z;
	int i = bdx * bx + tx; int j = bdy * by + ty; int k = bdz * bz + tz;
	int p = INDEX(k,j,i,w);
	if (j >= w || i >= w || k >= w) return;
	float ksq = i*i + j*j + k*k;
	float vol = 4.0*3.1415926*ks*ks*ks/3.0;
	if (ksq < ks*ks)
	{
		filter[p] = 1./vol;
	}
	else
	{
		filter[p] = 0;
	}
 }

__global__ void update_kernel(float* ionized, float* smoothed, const int w, float R, float S0)
{
  int tx = threadIdx.x;  int ty = threadIdx.y; int tz = threadIdx.z;
  int bx = blockIdx.x;   int by = blockIdx.y; int bz = blockIdx.z;
  int bdx = blockDim.x;  int bdy = blockDim.y; int bdz = blockDim.z;
  int i = bdx * bx + tx; int j = bdy * by + ty; int k = bdz * bz + tz;
  int p = INDEX(k,j,i,w);
  if (j >= w || i >= w || k >= w || ionized[p] == 1) return;

  float delta0 = smoothed[p];
  float deltac = %(DELTAC)s;
  float smin = %(SMIN)s;
  float fcoll = 1 - erf((deltac - delta0)/sqrt(2*(smin - S0)));
  ionized[p] = fcoll;
  //if (fcoll >= 1/%(ZETA)s) ionized[p] = 1.0;
  //else if (R==%(RMIN)s) { ionized[p] = fcoll * %(ZETA)s; }
 }